#include "hip/hip_runtime.h"

#include <iostream>

#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
        exit(-1); \
    }

__global__ void matmul_elem(int n, float* a, float* b, float* c)
{
	int column = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (row < n && column < n)
	{
		float dot_prod = 0.f;
		for (int i = 0; i < n; i++)
		{
			dot_prod += a[row * n + i] * b[i * n + column];
		}
		c[row * n + column] = dot_prod;
	}
}

int main() {
	float *a_h, *b_h, *c_h;
	float *a_d, *b_d, *c_d;
	int N = 3;

	a_h = new float[N * N];
	b_h = new float[N * N];
	c_h = new float[N * N];

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			a_h[i * N + j] = float(i * N + j);
			b_h[i * N + j] = float(i * N + j);
		}
	}

	CUDA_CHECK(hipMalloc((void**) &a_d, N*N*sizeof(float)));
	CUDA_CHECK(hipMalloc((void**) &b_d, N*N*sizeof(float)));
	CUDA_CHECK(hipMalloc((void**) &c_d, N*N*sizeof(float)));

	CUDA_CHECK(hipMemcpy(a_d, a_h, N*N*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(b_d, b_h, N*N*sizeof(float), hipMemcpyHostToDevice));

	dim3 blockSize(16, 16);
	dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);
	matmul_elem<<<gridSize, blockSize>>>(N, a_d, b_d, c_d);
	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(c_h, c_d, N*N*sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			std::cout << int(c_h[i * N + j]) << ' ';
		}
		std::cout << std::endl;
	}

	hipFree(a_d);
	hipFree(b_d);
	hipFree(c_d);

	delete[] a_h;
	delete[] b_h;
	delete[] c_h;
}